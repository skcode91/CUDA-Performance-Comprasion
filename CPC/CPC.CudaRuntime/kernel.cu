#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "../CPC.Common/Helpers/MatrixHelper.cpp"
#include "../CPC.Common/Helpers/CalculationHelper.h"
#include "../CPC.Common/Helpers/BinaryFileHelper.cpp"

__device__ inline double getMedian(double* values, int size)
{
	for (int i = 1; i < size; i++) {
		double key = values[i];
		int j = i - 1;
		while (j >= 0 && values[j] > key) {
			values[j + 1] = values[j];
			j = j - 1;
		}
		values[j + 1] = key;
	}
	if (size % 2 == 0) {
		return (values[size / 2 - 1] + values[size / 2]) / 2.0;
	}
	else {
		return values[size / 2];
	}
}

__global__ void medianFilter(double* inputMatrix, double* outputMatrix, int sizeX, int sizeY)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int idx = iy * sizeX + ix;

	if (ix >= 1 && iy >= 1 && ix < sizeX - 1 && iy < sizeY - 1) {
		double windowValues[9];
		int windowIndex = 0;
		for (int dx = -1; dx <= 1; dx++) {
			for (int dy = -1; dy <= 1; dy++) {
				unsigned int windowIdx = (iy + dy) * sizeX + (ix + dx);
				windowValues[windowIndex++] = inputMatrix[windowIdx];
			}
		}
		outputMatrix[idx] = getMedian(windowValues, 9);
	}
}

__global__ void medianFilterFirstRow(double* inputMatrix, double* outputMatrix, int sizeX, int sizeY)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x + 1;
	unsigned int iy = 0;
	unsigned int idx = iy * sizeX + ix;

	if (ix < sizeX - 1) {
		double windowValues[6];
		int windowIndex = 0;
		for (int dx = -1; dx <= 1; dx++) {
			for (int dy = 0; dy <= 1; dy++) {
				unsigned int windowIdx = (iy + dy) * sizeX + (ix + dx);
				windowValues[windowIndex++] = inputMatrix[windowIdx];
			}
		}
		outputMatrix[idx] = getMedian(windowValues, 6);
	}
}

__global__ void medianFilterFirstColumn(double* inputMatrix, double* outputMatrix, int sizeX, int sizeY)
{
	unsigned int ix = 0;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y + 1;
	unsigned int idx = iy * sizeX + ix;

	if (iy < sizeY - 1) {
		double windowValues[6];
		int windowIndex = 0;
		for (int dx = 0; dx <= 1; dx++) {
			for (int dy = -1; dy <= 1; dy++) {
				unsigned int windowIdx = (iy + dy) * sizeX + (ix + dx);
				windowValues[windowIndex++] = inputMatrix[windowIdx];
			}
		}
		outputMatrix[idx] = getMedian(windowValues, 6);
	}
}

__global__ void medianFilterLastColumn(double* inputMatrix, double* outputMatrix, int sizeX, int sizeY)
{
	unsigned int ix = sizeX - 1;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y + 1;
	unsigned int idx = iy * sizeX + ix;

	if (iy < sizeY - 1) {
		double windowValues[6];
		int windowIndex = 0;
		for (int dx = -1; dx <= 0; dx++) {
			for (int dy = -1; dy <= 1; dy++) {
				unsigned int windowIdx = (iy + dy) * sizeX + (ix + dx);
				windowValues[windowIndex++] = inputMatrix[windowIdx];
			}
		}
		outputMatrix[idx] = getMedian(windowValues, 6);
	}
}


__global__ void medianFilterLastRow(double* inputMatrix, double* outputMatrix, int sizeX, int sizeY)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x + 1;
	unsigned int iy = sizeY - 1;
	unsigned int idx = iy * sizeX + ix;

	if (ix < sizeX - 1) {
		double windowValues[6];
		int windowIndex = 0;
		for (int dx = -1; dx <= 1; dx++) {
			for (int dy = -1; dy <= 0; dy++) {
				unsigned int windowIdx = (iy + dy) * sizeX + (ix + dx);
				windowValues[windowIndex++] = inputMatrix[windowIdx];
			}
		}
		outputMatrix[idx] = getMedian(windowValues, 6);
	}
}

__global__ void calculateCorners(double* inputMatrix, double* outputMatrix, int sizeX, int sizeY)
{
	double cornerValues[4];


	// left - up corner
	cornerValues[0] = inputMatrix[0];
	cornerValues[1] = inputMatrix[1];
	cornerValues[2] = inputMatrix[sizeX];
	cornerValues[3] = inputMatrix[sizeX + 1];
	outputMatrix[0] = getMedian(cornerValues, 4);

	// right - up corner
	cornerValues[0] = inputMatrix[sizeX - 2];
	cornerValues[1] = inputMatrix[sizeX - 1];
	cornerValues[2] = inputMatrix[2 * sizeX - 2];
	cornerValues[3] = inputMatrix[2 * sizeX - 1];
	outputMatrix[sizeX - 1] = getMedian(cornerValues, 4);

	// left - down corner
	cornerValues[0] = inputMatrix[(sizeY - 2) * sizeX];
	cornerValues[1] = inputMatrix[(sizeY - 2) * sizeX + 1];
	cornerValues[2] = inputMatrix[(sizeY - 1) * sizeX];
	cornerValues[3] = inputMatrix[(sizeY - 1) * sizeX + 1];
	outputMatrix[(sizeY - 1) * sizeX] = getMedian(cornerValues, 4);

	// right - down corner
	cornerValues[0] = inputMatrix[sizeX * (sizeY - 1) - 2];
	cornerValues[1] = inputMatrix[sizeX * (sizeY - 1) - 1];
	cornerValues[2] = inputMatrix[sizeX * sizeY - 2];
	cornerValues[3] = inputMatrix[sizeX * sizeY - 1];
	outputMatrix[sizeX * sizeY - 1] = getMedian(cornerValues, 4);
}

int main()
{
		//	//	//	settings	//	//	//	//	//	//

	const int sizeX = 19000;	// cols
	const int sizeY = 19000;	// rows

	const std::string matrixFilePath = "D:/matrix.bin";
	const std::string probeFilePath = "D:/probe.txt";

	//	//	//	//	//	//	//	//	//	//	//	//

	std::cout << "Ilosc pamieci wykorzystywanej przez macierz to " << sizeY * sizeX / 1024 / 1024 * sizeof(double) << " Mb" << std::endl;

	// allocate memory

	double* inputMatrixVector = new double[sizeX * sizeY];
	double** inputMatrix = new double* [sizeX];
	for (int i = 0; i < sizeX; i++)
	{
		inputMatrix[i] = inputMatrixVector + i * sizeY;
	}

	double* outputMatrixVector = new double[sizeX * sizeY];
	double** outputMatrix = new double* [sizeX];
	for (int i = 0; i < sizeX; i++)
	{
		outputMatrix[i] = outputMatrixVector + i * sizeY;
	}


	std::cout << "Otwieranie pliku..." << std::endl;
	CPC::Common::Helpers::BinaryFileHelper::readMatrixFromFile(inputMatrix, matrixFilePath, sizeY, sizeX);


	// allocate memory on GPU
	double* gpuOutputMatrix;
	double* gpuInputMatrix;


	hipMalloc(&gpuInputMatrix, sizeX * sizeY * sizeof(double));
	hipMalloc(&gpuOutputMatrix, sizeX * sizeY * sizeof(double));

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("hipMemcpy error 1: %s\n", hipGetErrorString(error));
	}
	// copy from host to gpu
	hipMemcpy(gpuInputMatrix, inputMatrixVector, sizeX * sizeY * sizeof(double), hipMemcpyHostToDevice);
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("hipMemcpy error 1: %s\n", hipGetErrorString(error));
	}

	// kernel sizes
	int blockSizeX = 32;
	int blockSizeY = 32;


	dim3 blockMedian(blockSizeX, blockSizeY);
	dim3 gridMedian((sizeX + blockMedian.x - 1) / blockMedian.x, (sizeY + blockMedian.y - 1) / blockMedian.y);

	dim3 blockFirstRow(blockSizeX);
	dim3 gridFirstRow((sizeX - 2 + blockFirstRow.x - 1) / blockFirstRow.x);

	dim3 blockFirstColumn(1, blockSizeY);
	dim3 gridFirstColumn((sizeY - 2 + blockFirstColumn.y - 1) / blockFirstColumn.y);

	dim3 blockLastColumn(1, blockSizeY);
	dim3 gridLastColumn((sizeY - 2 + blockLastColumn.y - 1) / blockLastColumn.y);

	dim3 blockLastRow(blockSizeX, 1);
	dim3 gridLastRow((sizeX - 2 + blockLastRow.x - 1) / blockLastRow.x);

	// computing 
	clock_t start = clock();

	medianFilter << <gridMedian, blockMedian >> > (gpuInputMatrix, gpuOutputMatrix, sizeX, sizeY);

	hipDeviceSynchronize();

	medianFilterFirstRow << <gridFirstRow, blockFirstRow >> > (gpuInputMatrix, gpuOutputMatrix, sizeX, sizeY);
	medianFilterLastRow << <gridLastRow, blockLastRow >> > (gpuInputMatrix, gpuOutputMatrix, sizeX, sizeY);
	medianFilterFirstColumn << <gridFirstColumn, blockFirstColumn >> > (gpuInputMatrix, gpuOutputMatrix, sizeX, sizeY);
	medianFilterLastColumn << <gridLastColumn, blockLastColumn >> > (gpuInputMatrix, gpuOutputMatrix, sizeX, sizeY);
	calculateCorners << <1, 1 >> > (gpuInputMatrix, gpuOutputMatrix, sizeX, sizeY);

	hipDeviceSynchronize();

	clock_t end = clock();

	// end of computing


	double duration = double(end - start) / CLOCKS_PER_SEC * 500;

	std::cout << "Obliczenia zakonczono w czasie " << duration << " ms" << std::endl;

	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("hipMemcpy error 4: %s\n", hipGetErrorString(error));
	}

	hipDeviceSynchronize();

	hipMemcpy(outputMatrixVector, gpuOutputMatrix, sizeX * sizeY * sizeof(double), hipMemcpyDeviceToHost);


	//std::cout << std::endl;
	//printMatrix(outputMatrix[0], sizeX, sizeY);
	//std::cout << std::endl;
	//printMatrix(inputMatrix[0], sizeX, sizeY);

	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("hipMemcpy error (DeviceToHost): %s\n", hipGetErrorString(error));
	}
	CPC::Common::Helpers::BinaryFileHelper::validateMatrixProbe(probeFilePath, outputMatrix, sizeY, sizeX);

	hipFree(gpuInputMatrix);
	hipFree(gpuOutputMatrix);

	return 0;
}
